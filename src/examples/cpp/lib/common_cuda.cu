#include "common.h"

#ifdef __cplusplus
extern "C" {
#endif

int getNumCUDADevices() {
    int ndevices;
    hipError_t err = hipGetDeviceCount(&ndevices);
    if (err == hipErrorNoDevice) {
        return 0;
    } else if (err == hipSuccess) {
        return ndevices;
    } else {
        fprintf(stderr, "Unexpected error in getNumCUDADevices: %s\n",
                hipGetErrorString(err));
        exit(1);
    }
}

#ifdef __cplusplus
}
#endif
