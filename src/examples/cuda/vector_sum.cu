
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK(call) { \
    hipError_t error; \
    if ((error = (call)) != hipSuccess) { \
        fprintf(stderr, "Error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(error)); \
        exit(1); \
    } \
}

__global__ void kernel(int *A, int *B, int *C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        C[tid] = A[tid] + B[tid];
    }
}

int main(int argc, char **argv) {
    int i;
    int N = 1024 * 1024;

    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    h_A = (int *)malloc(sizeof(int) * N);
    h_B = (int *)malloc(sizeof(int) * N);
    h_C = (int *)malloc(sizeof(int) * N);

    CHECK(hipMalloc((void **)&d_A, sizeof(int) * N));
    CHECK(hipMalloc((void **)&d_B, sizeof(int) * N));
    CHECK(hipMalloc((void **)&d_C, sizeof(int) * N));

    for (i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    CHECK(hipMemcpy(d_A, h_A, sizeof(int) * N, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(d_B, h_B, sizeof(int) * N, hipMemcpyHostToDevice))

    int threads_per_block = 128;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    kernel<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_C, N);

    CHECK(hipMemcpy(h_C, d_C, sizeof(int) * N, hipMemcpyDeviceToHost));

    FILE *fp = fopen("dump.out", "w");
    for (i = 0; i < N; i++) {
        fprintf(fp, "%d\n", h_C[i]);
    }
    fclose(fp);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    return 0;
}
