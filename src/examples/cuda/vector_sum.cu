#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "checkpoint.h"

#define CHECK(call) { \
    if ((error = (call)) != hipSuccess) { \
        fprintf(stderr, "Error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(error)); \
        exit(1); \
    } \
}

__global__ void kernel(int *A, int *B, int *C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        C[tid] = A[tid] + B[tid];
    }
}

int main(int argc, char **argv) {
    hipError_t error;
    int i;
    int N = 1024 * 1024;

    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    h_A = (int *)malloc(sizeof(int) * N);
    h_B = (int *)malloc(sizeof(int) * N);
    h_C = (int *)malloc(sizeof(int) * N);

    hipMalloc((void **)&d_A, sizeof(int) * N);
    hipMalloc((void **)&d_B, sizeof(int) * N);
    hipMalloc((void **)&d_C, sizeof(int) * N);

    for (i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    hipMemcpy(d_A, h_A, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(int) * N, hipMemcpyHostToDevice);

    int threads_per_block = 128;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    kernel<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, sizeof(int) * N, hipMemcpyDeviceToHost);

    checkpoint();

    FILE *fp = fopen("dump.out", "w");
    for (i = 0; i < N; i++) {
        fprintf(fp, "%d\n", h_C[i]);
    }
    fclose(fp);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
