#include "hip/hip_runtime.h"
#include "type_info.h"
#include "heap_allocation.h"
#include "ptr_and_size.h"
#include "chimes_common.h"
#include "heap_tree.h"
#include <map>
#include <cstdarg>

using namespace std;

extern void malloc_impl(void *new_ptr, size_t nbytes, size_t group,
        int is_cuda_alloc, int is_ptr, int is_struct, int elem_size,
        int *ptr_field_offsets, int n_ptr_field_offsets, bool filled);
extern heap_allocation *free_impl(const void *ptr, heap_allocation *alloc);
extern map<void *, heap_allocation *>::iterator find_in_heap(void *ptr);

__global__ void translate_pointers_kernel(void *arr, int nelems, int elem_size,
        int *ptr_offsets, int n_ptr_offsets, void **old_ptrs, void **new_ptrs,
        size_t *ptr_size, int n_translations);

void cudaMalloc_helper(hipError_t err, void **ptr, size_t size, size_t group,
        int is_ptr, int is_struct, ...) {
    if (err != hipSuccess) {
        return;
    }

    chimes_type_info info; memset(&info, 0x00, sizeof(info));
    if (is_struct) {
        va_list vl;
        va_start(vl, is_struct);
        parse_type_info(&vl, &info);
        va_end(vl);
    }

    malloc_impl(*ptr, size, group, 1, is_ptr, is_struct, info.elem_size,
            info.ptr_field_offsets, info.n_ptr_fields, false);
}

void cudaFree_helper(hipError_t err, void *ptr, size_t group) {
    if (err == hipSuccess) {
        free_impl(ptr);
    }
}

void translate_cuda_pointers(void *d_arr, int nelems, int elem_size,
        int *ptr_offsets, unsigned n_ptr_offsets, heap_tree *old_to_new) {

    int *d_ptr_offsets;
    void **d_old, **d_new;
    size_t *d_size;

    void **h_old, **h_new;
    size_t *h_size;
    const int index = old_to_new->serialize(&h_old, &h_new, &h_size);

    CHECK(hipMalloc((void **)&d_ptr_offsets, sizeof(int) * n_ptr_offsets));
    CHECK(hipMalloc((void **)&d_old, sizeof(void *) * index));
    CHECK(hipMalloc((void **)&d_new, sizeof(void *) * index));
    CHECK(hipMalloc((void **)&d_size, sizeof(size_t) * index));

    CHECK(hipMemcpy(d_ptr_offsets, ptr_offsets,
                sizeof(int) * n_ptr_offsets, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_old, h_old, sizeof(void *) * index, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_new, h_new, sizeof(void *) * index, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_size, h_size, sizeof(size_t) * index, hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (nelems + threads - 1) / threads;
    translate_pointers_kernel<<<blocks, threads>>>(d_arr, nelems, elem_size,
            d_ptr_offsets, n_ptr_offsets, d_old, d_new, d_size, index);
    CHECK(hipDeviceSynchronize());

    CHECK(hipFree(d_ptr_offsets));
    CHECK(hipFree(d_old));
    CHECK(hipFree(d_new));
    CHECK(hipFree(d_size));
    free(h_old);
    free(h_new);
    free(h_size);
}

__device__ void *translate_ptr(void *ptr, void **old_ptrs, void **new_ptrs,
        size_t *ptr_size, int n_translations) {
    int i;
    unsigned char *c_ptr = (unsigned char *)ptr;
    for (i = 0; i < n_translations; i++) {
        unsigned char *old = (unsigned char *)old_ptrs[i];
        size_t size = ptr_size[i];
        size_t diff = c_ptr - old;
        if (diff < size) {
            return (void *)(((unsigned char *)(new_ptrs[i])) + diff);
        }
    }
    return NULL;
}

__global__ void translate_pointers_kernel(void *arr, int nelems, int elem_size,
        int *ptr_offsets, int n_ptr_offsets, void **old_ptrs, void **new_ptrs,
        size_t *ptr_size, int n_translations) {
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= nelems) return;

    unsigned char *elem_ptr = (unsigned char *)arr + (tid * elem_size);
    for (i = 0; i < n_ptr_offsets; i++) {
        void **ptr_ptr = (void **)(elem_ptr + ptr_offsets[i]);
        void *new_ptr = translate_ptr(*ptr_ptr, old_ptrs, new_ptrs, ptr_size, n_translations);
        *ptr_ptr = new_ptr;
    }
}
