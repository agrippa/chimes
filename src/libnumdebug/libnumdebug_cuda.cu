#include "type_info.h"
#include <cstdarg>

extern void malloc_helper(void *new_ptr, size_t nbytes, int group,
        int is_cuda_alloc, int has_type_info, int is_ptr, int is_struct,
        int elem_size, int *ptr_field_offsets, int n_ptr_field_offsets);

hipError_t cudaMalloc_wrapper(void **ptr, size_t size, int group,
        int has_type_info, ...) {
    hipError_t err = hipMalloc(ptr, size);
    if (err != hipSuccess) {
        return err;
    }

    numdebug_type_info info; memset(&info, 0x00, sizeof(info));
    if (has_type_info) {
        va_list vl;
        va_start(vl, has_type_info);
        parse_type_info(vl, &info);
        va_end(vl);
    }

    malloc_helper(*ptr, size, group, 1, has_type_info, info.is_ptr,
            info.is_struct, info.elem_size, info.ptr_field_offsets,
            info.n_ptr_fields);

    return hipSuccess;
}

hipError_t cudaFree_wrapper(void *ptr, int group) {
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        return err;
    }

    return hipSuccess;
}
