#include "hip/hip_runtime.h"
#include "checkpoint.h"
#include <assert.h>
#include <stdio.h>

__global__ void kernel(int *arr) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    arr[tid] = tid;
}

int main(int argc, char **argv) {
    int *d_arr;
    int *h_arr = (int *)malloc(sizeof(int) * 100);
    hipMalloc((void **)&d_arr, sizeof(int) * 100);
    hipMemcpy(d_arr, h_arr, sizeof(int) * 100, hipMemcpyHostToDevice);
    h_arr[0] = 0;
    h_arr[1] = 1;
    h_arr[2] = 2;

    checkpoint();

    kernel<<<10, 128>>>(d_arr);

    assert(h_arr != NULL);
    assert(d_arr != NULL);

    assert(h_arr[0] == 0);
    assert(h_arr[1] == 1);
    assert(h_arr[2] == 2);

    hipMemcpy(h_arr, d_arr, sizeof(int) * 100, hipMemcpyDeviceToHost);

    assert(h_arr[0] == 0);
    assert(h_arr[1] == 1);
    assert(h_arr[2] == 2);

    hipFree(d_arr);
    free(h_arr);
    return 0;
}
