#include "hip/hip_runtime.h"
#include "checkpoint.h"
#include <assert.h>
#include <stdio.h>

__global__ void kernel(int *arr) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    arr[tid] = tid;
}

int main(int argc, char **argv) {
    int *d_arr;
    int *h_arr = (int *)malloc(sizeof(int) * 100);
    hipMalloc((void **)&d_arr, sizeof(int) * 100);
    hipMemcpy(d_arr, h_arr, sizeof(int) * 100, hipMemcpyHostToDevice);
    h_arr[0] = 0;

    checkpoint();

    kernel<<<10, 128>>>(d_arr);

    assert(h_arr != NULL);

    hipFree(d_arr);
    free(h_arr);
    return 0;
}
