#include "hip/hip_runtime.h"
#include "checkpoint.h"
#include <assert.h>
#include <stdio.h>

#define CHECK(call) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
        exit(1); \
    } \
}

__global__ void kernel(int *arr) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    arr[tid] = tid;
}

int main(int argc, char **argv) {
    int *d_arr;
    int *h_arr = (int *)malloc(sizeof(int) * 100);
    CHECK(hipMalloc((void **)&d_arr, sizeof(int) * 100));
    CHECK(hipMemcpy(d_arr, h_arr, sizeof(int) * 100, hipMemcpyHostToDevice));
    h_arr[0] = 0;
    h_arr[1] = 1;
    h_arr[2] = 2;

    checkpoint();

    kernel<<<10, 128>>>(d_arr);

    assert(h_arr != NULL);
    assert(d_arr != NULL);

    assert(h_arr[0] == 0);
    assert(h_arr[1] == 1);
    assert(h_arr[2] == 2);

    CHECK(hipMemcpy(h_arr, d_arr, sizeof(int) * 100, hipMemcpyDeviceToHost));

    assert(h_arr[0] == 0);
    assert(h_arr[1] == 1);
    assert(h_arr[2] == 2);

    CHECK(hipFree(d_arr));
    free(h_arr);
    return 0;
}
