#include "checkpoint.h"
#include <assert.h>
#include <stdio.h>

int main(int argc, char **argv) {
    int *d_arr;
    int *h_arr = (int *)malloc(sizeof(int) * 100);
    hipMalloc((void **)&d_arr, sizeof(int) * 100);
    hipMemcpy(d_arr, h_arr, sizeof(int) * 100, hipMemcpyHostToDevice);
    h_arr[0] = 0;

    checkpoint();

    assert(h_arr != NULL);

    hipFree(d_arr);
    free(h_arr);
    return 0;
}
