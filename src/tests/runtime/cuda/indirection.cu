#include "hip/hip_runtime.h"
#include <stdio.h>
#include "checkpoint.h"
#include <assert.h>

#define CHECK(call) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
        exit(1); \
    } \
}

__global__ void kernel(int **ptr_to_ptr, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int *ptr = *ptr_to_ptr;
    if (tid < N) {
        ptr[tid] = tid;
    }
}

int main(int argc, char **argv) {
    int i;
    int N = 1024;
    int *d_arr, *h_arr;
    int **d_ptr_to_ptr, *h_ptr_to_ptr;
    int *d_tmp_buffer;

    h_arr = (int *)malloc(sizeof(int) * N);
    CHECK(hipMalloc((void **)&d_ptr_to_ptr, sizeof(int *)));
    CHECK(hipMalloc((void **)&d_tmp_buffer, sizeof(int) * N));
    CHECK(hipMalloc((void **)&d_arr, sizeof(int) * N));
    CHECK(hipFree(d_tmp_buffer));

    checkpoint();

    CHECK(hipMemcpy(d_ptr_to_ptr, &d_arr, sizeof(int *), hipMemcpyHostToDevice));

    wait_for_checkpoint();
    checkpoint();

    kernel<<<4, 256>>>(d_ptr_to_ptr, N);
    CHECK(hipDeviceSynchronize());

    wait_for_checkpoint();
    checkpoint();

    CHECK(hipMemcpy(h_arr, d_arr, sizeof(int) * N, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&h_ptr_to_ptr, d_ptr_to_ptr, sizeof(void *), hipMemcpyDeviceToHost));

    assert(h_ptr_to_ptr == d_arr);
    for (i = 0; i < N; i++) {
        assert(h_arr[i] == i);
    }

    CHECK(hipFree(d_ptr_to_ptr));
    CHECK(hipFree(d_arr));
    free(h_arr);

    return 0;
}
