#include "hip/hip_runtime.h"
#include "checkpoint.h"
#include <assert.h>
#include <stdio.h>

#define CHECK(call) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
        exit(1); \
    } \
}

__global__ void vector_add(int *A, int *B, int *C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        C[tid] = A[tid] + B[tid];
    }
}

int main(int argc, char **argv) {
    int i;
    int N = 1024;
    int iter, iters = 10;
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    h_A = (int *)malloc(N * sizeof(int));
    h_B = (int *)malloc(N * sizeof(int));
    h_C = (int *)malloc(N * sizeof(int));

    CHECK(hipMalloc((void **)&d_A, N * sizeof(int)));
    CHECK(hipMalloc((void **)&d_B, N * sizeof(int)));
    CHECK(hipMalloc((void **)&d_C, N * sizeof(int)));

    for (i = 0; i < N; i++) {
        h_C[i] = i;
    }

    for (iter = 0; iter < iters; iter++) {
        for (i = 0; i < N; i++) {
            h_A[i] = h_C[i];
            h_B[i] = 2 * h_C[i];
        }

        CHECK(hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_B, h_B, N * sizeof(int), hipMemcpyHostToDevice));

        wait_for_checkpoint();
        checkpoint();

        vector_add<<<4, 256>>>(d_A, d_B, d_C, N);

        CHECK(hipMemcpy(h_C, d_C, N * sizeof(int), hipMemcpyDeviceToHost));

        for (i = 0; i < N; i++) {
            assert(h_C[i] == h_A[i] + h_B[i]);
        }
    }

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
